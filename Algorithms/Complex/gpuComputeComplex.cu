#include "hip/hip_runtime.h"
/*
 * This file is part of the SUperman repository: https://github.com/SU-HPC/SUPerman
 * Author(s): Deniz Elbek, Fatih Taşyaran, Bora Uçar, and Kamer Kaya.
 *
 * Please see the papers:
 * 
 * @article{Elbek2025SUperman,
 *   title   = {SUperman: Efficient Permanent Computation on GPUs},
 *   author  = {Elbek, Deniz and Taşyaran, Fatih and Uçar, Bora and Kaya, Kamer},
 *   journal = {arXiv preprint arXiv:2502.16577},
 *   year    = {2025},
 *   doi     = {10.48550/arXiv.2502.16577},
 *   url     = {https://arxiv.org/abs/2502.16577}
 * }
 *
 * @article{Elbek2025FullyAutomated,
 *   title   = {Fully-Automated Code Generation for Efficient Computation of Sparse Matrix Permanents on GPUs},
 *   author  = {Elbek, Deniz and Kaya, Kamer},
 *   journal = {arXiv preprint arXiv:2501.15126},
 *   year    = {2025},
 *   doi     = {10.48550/arXiv.2501.15126},
 *   url     = {https://arxiv.org/abs/2501.15126}
 * }
 */

#ifndef SUPERMAN_GPUCOMPUTECOMPLEX_CU
#define SUPERMAN_GPUCOMPUTECOMPLEX_CU

#include "Matrix.h"
#include "Settings.h"
#include <complex>
#include "hip/hip_runtime.h"
#include "GPUHelpers.cuh"
#include "Helpers.h"
#include "hip/hip_complex.h"
#include <iomanip>
#include "omp.h"


__global__ void xRegisterMSharedComplex(hipDoubleComplex* mat,
                                        hipDoubleComplex* x,
                                        hipDoubleComplex* p,
                                        int nov,
                                        long long start,
                                        long long end,
                                        long long chunkSize)
{
    int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
    int totalThreadCount = gridDim.x * blockDim.x;

    hipDoubleComplex myResult = make_hipDoubleComplex(0, 0);

    #define REG(reg, number) hipDoubleComplex reg;
        REGISTERS
    #undef REG

    extern __shared__ char sharedMemory[];
    hipDoubleComplex* sharedMat = (hipDoubleComplex*)sharedMemory;

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < nov; ++i)
        {
            for (int j = 0; j < nov; ++j)
            {
                sharedMat[i * nov + j] = mat[i * nov + j];
            }
        }
    }

    __syncthreads();

    #define REG(reg, number) if (number < nov) {reg = x[number];}
        REGISTERS
    #undef REG

    if (chunkSize == -1)
    {
        chunkSize = (end - start + totalThreadCount - 1) / totalThreadCount;
    }
    long long myStart = start + (threadID * chunkSize);
    long long myEnd = min(start + ((threadID + 1) * chunkSize), end);

    long long gray = (myStart - 1) ^ ((myStart - 1) >> 1); // gray code for the previous subset
    // getting the x vector from the previous subset
    for (int j = 0; j < (nov - 1); ++j)
    {
        if ((gray >> j) & 1LL) // was jth column included?
        {
            #define REG(reg, number) if (number < nov) {reg = hipCadd(reg, sharedMat[j * nov + number]);}
                REGISTERS
            #undef REG
        }
    }

    // are we starting with a negative product sign?
    hipDoubleComplex productSign = (myStart & 1LL) ? make_hipDoubleComplex(-1, 0) : make_hipDoubleComplex(1, 0);

    for (long long i = myStart; i < myEnd; ++i)
    {
        long long grayDifference = (i ^ (i >> 1)) ^ gray;
        int columnChanged = __ffsll(grayDifference) - 1; // column no that was added or removed
        gray ^= (1LL << columnChanged);

        // is column removed or added
        hipDoubleComplex added = ((1LL << columnChanged) & gray) ? make_hipDoubleComplex(1, 0) : make_hipDoubleComplex(-1, 0);

        hipDoubleComplex product = make_hipDoubleComplex(1, 0);
        #define REG(reg, number) if (number < nov) {reg = hipCadd(reg, hipCmul(added, sharedMat[columnChanged * nov + number])); product = hipCmul(product, reg);}
            REGISTERS
        #undef REG

        myResult = hipCadd(myResult, hipCmul(productSign, product));
        productSign = hipCmul(productSign, make_hipDoubleComplex(-1, 0));
    }

    p[threadID] = hipCadd(p[threadID], myResult);
}

__global__ void xRegisterMSharedComplexMatSpecific(hipDoubleComplex* mat,
                                        hipDoubleComplex* x,
                                        hipDoubleComplex* p,
                                        int nov,
                                        long long start,
                                        long long end,
                                        long long chunkSize)
{
    int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
    int totalThreadCount = gridDim.x * blockDim.x;

    hipDoubleComplex myResult = make_hipDoubleComplex(0, 0);

    #define REG(reg, number) hipDoubleComplex reg;
        SPECIFIC
    #undef REG

    extern __shared__ char sharedMemory[];
    hipDoubleComplex* sharedMat = (hipDoubleComplex*)sharedMemory;

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < nov; ++i)
        {
            for (int j = 0; j < nov; ++j)
            {
                sharedMat[i * nov + j] = mat[i * nov + j];
            }
        }
    }

    __syncthreads();

    #define REG(reg, number) {reg = x[number];}
        SPECIFIC
    #undef REG

    if (chunkSize == -1)
    {
        chunkSize = (end - start + totalThreadCount - 1) / totalThreadCount;
    }
    long long myStart = start + (threadID * chunkSize);
    long long myEnd = min(start + ((threadID + 1) * chunkSize), end);

    long long gray = (myStart - 1) ^ ((myStart - 1) >> 1); // gray code for the previous subset
    // getting the x vector from the previous subset
    for (int j = 0; j < (nov - 1); ++j)
    {
        if ((gray >> j) & 1LL) // was jth column included?
        {
            #define REG(reg, number) {reg = hipCadd(reg, sharedMat[j * nov + number]);}
                SPECIFIC
            #undef REG
        }
    }

    // are we starting with a negative product sign?
    hipDoubleComplex productSign = (myStart & 1LL) ? make_hipDoubleComplex(-1, 0) : make_hipDoubleComplex(1, 0);

    for (long long i = myStart; i < myEnd; ++i)
    {
        long long grayDifference = (i ^ (i >> 1)) ^ gray;
        int columnChanged = __ffsll(grayDifference) - 1; // column no that was added or removed
        gray ^= (1LL << columnChanged);

        // is column removed or added
        hipDoubleComplex added = ((1LL << columnChanged) & gray) ? make_hipDoubleComplex(1, 0) : make_hipDoubleComplex(-1, 0);

        hipDoubleComplex product = make_hipDoubleComplex(1, 0);
        #define REG(reg, number) {reg = hipCadd(reg, hipCmul(added, sharedMat[columnChanged * nov + number])); product = hipCmul(product, reg);}
            SPECIFIC
        #undef REG

        myResult = hipCadd(myResult, hipCmul(productSign, product));
        productSign = hipCmul(productSign, make_hipDoubleComplex(-1, 0));
    }

    p[threadID] = hipCadd(p[threadID], myResult);
}

void gpuComputeComplex(Matrix<std::complex<double>>* matrix, Settings* settings)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, settings->deviceID);

    gpuErrchk( hipSetDevice(settings->deviceID) )

    int nov = matrix->nov;
#ifdef MAT_SPECIFIC_COMPILATION
    if (NOV != nov)
    {
        throw std::runtime_error("It seems that you have made a matrix specific compilation but the size of the matrix does not match with that of your indicated size during compilation. Perhaps decomposition reduced the size on the runtime?\n");
    }
#endif
    hipDoubleComplex* mat = new hipDoubleComplex[nov * nov];
    for (int i = 0; i < nov; ++i)
    {
        for (int j = 0; j < nov; ++j)
        {
            mat[i * nov + j] = make_hipDoubleComplex(matrix->mat[i * nov + j].real(), matrix->mat[i * nov + j].imag());
        }
    }

    hipDoubleComplex x[nov];
    hipDoubleComplex product = make_hipDoubleComplex(1.0, 0.0);
    for (int i = 0; i < nov; ++i)
    {
        hipDoubleComplex rowSum = make_hipDoubleComplex(0.0, 0.0);
        for (int j = 0; j < nov; ++j)
        {
            rowSum = hipCadd(rowSum, mat[i * nov + j]);
        }
        x[i] = hipCsub(mat[i * nov + (nov - 1)], hipCdiv(rowSum, make_hipDoubleComplex(2.0, 0.0)));
        product = hipCmul(product, x[i]);
    }
    hipDoubleComplex productSum = product;

    hipDoubleComplex* matTransposed = new hipDoubleComplex[nov * nov];
    for (int i = 0; i < nov; ++i)
    {
        for (int j = 0; j < nov; ++j)
        {
            matTransposed[j * nov + i] = mat[i * nov + j];
        }
    }

    int gridDim;
    int blockDim;
    V = nov;
    gpuErrchk( hipOccupancyMaxPotentialBlockSizeVariableSMem(
            &gridDim,
            &blockDim,
            xRegisterMSharedComplex,
            dpMSharedComplex,
            0) )

    int noSM = prop.multiProcessorCount;
    int sharedMemoryPerBlock = dpMSharedComplex(blockDim);
    int maxSharedMemoryPerBlock= prop.sharedMemPerBlock;
    int maxSharedMemoryPerSM = prop.sharedMemPerMultiprocessor;
    int maxRegsPerSM = prop.regsPerMultiprocessor;
    int totalThreadCount = gridDim * blockDim;

    int maxBlocks;
    gpuErrchk( hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &maxBlocks,
            xRegisterMSharedComplex,
            blockDim,
            sharedMemoryPerBlock
    ) )

#ifndef SILENT
#pragma omp critical
    {
        static bool printed = false;
        if (!printed)
        {
            std::cout << "Permanent is being computed on device id: " << settings->deviceID << ", " << prop.name << std::endl;
            std::cout << "Matrix Size: " << (nov * nov) * sizeof(hipDoubleComplex) << " bytes" << std::endl;
            std::cout << "X Vector Size: " << (nov * sizeof(hipDoubleComplex)) << " bytes" << std::endl;
            std::cout << "Number of streaming multiprocessors: " << noSM << std::endl;
            std::cout << "Shared memory used per block: " << sharedMemoryPerBlock << " bytes" << std::endl;
            std::cout << "Shared memory used per SM: " << (sharedMemoryPerBlock * maxBlocks) << " bytes" << std::endl;
            std::cout << (double(sharedMemoryPerBlock) / double(maxSharedMemoryPerBlock)) * 100 << "% of the entire shared memory dedicated per block is used" << std::endl;
            std::cout << ((double(sharedMemoryPerBlock) * maxBlocks) / double(maxSharedMemoryPerSM)) * 100 << "% of the entire shared memory dedicated per SM is used" << std::endl;
            std::cout << "Maximum number of registers that could be used per SM: " << maxRegsPerSM << std::endl;
            std::cout << "Grid Dimension: " << gridDim << std::endl;
            std::cout << "Block Dimension: " << blockDim << std::endl;
            std::cout << "Total number of threads: " << totalThreadCount << std::endl;
            std::cout << "Maximum number of blocks running concurrently on each SM: " << maxBlocks << std::endl;
            std::cout << "Maximum number of blocks running concurrently throughout the GPU: " << (maxBlocks * noSM) << std::endl;
            printed = true;
        }
    }
#endif

    double timeStart = omp_get_wtime();

    hipDoubleComplex* d_x;
    hipDoubleComplex* d_products;
    hipDoubleComplex* d_mat;

    gpuErrchk( hipMalloc(&d_x, nov * sizeof(hipDoubleComplex)) )
    gpuErrchk( hipMalloc(&d_products, totalThreadCount * sizeof(hipDoubleComplex)) )
    gpuErrchk( hipMemset(d_products, 0, totalThreadCount * sizeof(hipDoubleComplex)) )
    gpuErrchk( hipMalloc(&d_mat, (nov * nov) * sizeof(hipDoubleComplex)) )

    gpuErrchk( hipMemcpy(d_x, x, nov * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) )
    gpuErrchk( hipMemcpy(d_mat, matTransposed, (nov * nov) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) )

    hipDoubleComplex* h_products = new hipDoubleComplex[totalThreadCount];

    long long start = 1;
    long long end = (1LL << (nov - 1));
    long long left = (end - start);

    double current = 0;
    double next = 50;
    printProgressBar(current, settings->rank, settings->PID);

    while (totalThreadCount < left)
    {
        long long chunkSize = 1;
        while ((chunkSize * totalThreadCount) <= left)
        {
            chunkSize *= 2;
        }
        chunkSize /= 2;

        if (chunkSize == 1)
        {
            break;
        }

        #ifdef MAT_SPECIFIC_COMPILATION
        xRegisterMSharedComplexMatSpecific<<<gridDim, blockDim, sharedMemoryPerBlock>>>(
                d_mat,
                d_x,
                d_products,
                nov,
                start,
                end,
                chunkSize);
        #else
        xRegisterMSharedComplex<<<gridDim, blockDim, sharedMemoryPerBlock>>>(
                d_mat,
                d_x,
                d_products,
                nov,
                start,
                end,
                chunkSize);
        #endif

        gpuErrchk( hipDeviceSynchronize() )

        current += next;
        next /= 2;
        printProgressBar(current, settings->rank, settings->PID);

        long long thisIteration = totalThreadCount * chunkSize;
        left -= thisIteration;
        start += thisIteration;
    }

    #ifdef MAT_SPECIFIC_COMPILATION
    xRegisterMSharedComplexMatSpecific<<<gridDim, blockDim, sharedMemoryPerBlock>>>(
            d_mat,
            d_x,
            d_products,
            nov,
            start,
            end,
            -1);
    #else
    xRegisterMSharedComplex<<<gridDim, blockDim, sharedMemoryPerBlock>>>(
            d_mat,
            d_x,
            d_products,
            nov,
            start,
            end,
            -1);
    #endif

    gpuErrchk( hipDeviceSynchronize() )

    current = 100;
    printProgressBar(current, settings->rank, settings->PID);

    gpuErrchk( hipMemcpy( h_products, d_products, totalThreadCount * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) )

    for (int i = 0; i < totalThreadCount; ++i)
    {
        productSum = hipCadd(productSum, h_products[i]);
    }

    double timeEnd = omp_get_wtime();

    gpuErrchk( hipFree(d_x) )
    gpuErrchk( hipFree(d_products) )
    gpuErrchk( hipFree(d_mat) )

    delete[] mat;
    delete[] h_products;
    delete[] matTransposed;

    hipDoubleComplex result = hipCmul(make_hipDoubleComplex((4 * (nov % 2) - 2), 0), productSum);
    std::stringstream stream;
    stream << "Permanent: " << '(' << std::setprecision(settings->printingPrecision) << result.x << ',' << result.y << ')' << std::endl;
    stream << "Time took: " << timeEnd - timeStart << std::endl << std::endl;
    print(stream, settings->rank, settings->PID, -1);
}


#endif //SUPERMAN_GPUCOMPUTECOMPLEX_CU
