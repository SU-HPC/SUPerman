//
// Created by deniz on 5/24/24.
//

#include "GPUKernelWrappers.h"

// COMMON
#include "Permanent.h"
#include "DecomposePerman.h"
#include "Result.h"
#include "Settings.h"
#include "Matrix.h"
#include <stdexcept>
#include "SparseMatrix.h"
#include "hip/hip_runtime.h"

// SP
#include "SparseKernelDefinitions.cuh"
#include "spSingleGPU.cuh"
#include "spMultiGPU.cuh"

// DP
#include "DenseKernelDefinitions.cuh"
#include "dpSingleGPU.cuh"
#include "dpMultiGPU.cuh"

// Kernel Generation
#include "generatedKernels.cuh"
#include "KernelGenSingleGPU.cuh"
#include "KernelGenMultiGPU.cuh"

// MPI
#ifdef MPI_AVAILABLE
#include "spMultiGPUMPI.cuh"
#include "dpMultiGPUMPI.cuh"
#include "KernelGenMultiGPUMPI.cuh"
#endif


// SPARSE WRAPPERS
template <typename C, typename S>
extern Result gpuSPSingleGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xGlobalMGlobal, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xGlobalMShared, spMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if ((selectedAlgorithm == Algorithm::NAIVECODEGENERATION) || selectedAlgorithm == Algorithm::REGEFFICIENTCODEGENERATION)
    {
        auto permanent = new DecomposePerman<C, S, KernelGenSingleGPU<C, S, &globalKernel, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}

template <typename C, typename S>
extern Result gpuSPMultiGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xGlobalMGlobal, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xGlobalMShared, spMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if ((selectedAlgorithm == Algorithm::NAIVECODEGENERATION) || selectedAlgorithm == Algorithm::REGEFFICIENTCODEGENERATION)
    {
        auto permanent = new DecomposePerman<C, S, KernelGenMultiGPU<C, S, &globalKernel, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}

#ifdef MPI_AVAILABLE
template <typename C, typename S>
extern Result gpuSPMultiGPUMPI(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPUMPI<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPUMPI<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPUMPI<C, S, &SparseDefinitions::xGlobalMGlobal, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XGLOBALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPUMPI<C, S, &SparseDefinitions::xGlobalMShared, spMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if ((selectedAlgorithm == Algorithm::NAIVECODEGENERATION) || selectedAlgorithm == Algorithm::REGEFFICIENTCODEGENERATION)
    {
        auto permanent = new DecomposePerman<C, S, KernelGenMultiGPUMPI<C, S, &globalKernel, spNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}
#endif
// SPARSE WRAPPERS


// DENSE WRAPPERS
template <typename C, typename S>
extern Result gpuDPSingleGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedKahanMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #else
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMSharedKahan, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}

template <typename C, typename S>
extern Result gpuDPMultiGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedKahanMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #else
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedKahan, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}

#ifdef MPI_AVAILABLE
template <typename C, typename S>
extern Result gpuDPMultiGPUMPI(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedKahanMatSpecificCompilation, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #else
            if (settings->calculationPrecision == DD)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
            else if (settings->calculationPrecision == KAHAN)
            {
                auto permanent = new DecomposePerman<C, S, dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMSharedKahan, dpMShared<C, S> > >(matrix, *settings);
                result = permanent->computePermanentRecursively();
                delete permanent;
            }
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPUMPI<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPUMPI<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> > >(matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.\n");
    }

    return result;
}
#endif
// DENSE WRAPPERS


// FOR COMPILATION -SPARSE-
// Calculation: Double | Storage: Float
template extern Result gpuSPSingleGPU<double, float>(Matrix<float>* matrix, Settings* settings);
template extern Result gpuSPMultiGPU<double, float>(Matrix<float>* matrix, Settings* settings);
#ifdef MPI_AVAILABLE
template extern Result gpuSPMultiGPUMPI<double, float>(Matrix<float>* matrix, Settings* settings);
#endif

// Calculation: Double | Storage: Double
template extern Result gpuSPSingleGPU<double, double>(Matrix<double>* matrix, Settings* settings);
template extern Result gpuSPMultiGPU<double, double>(Matrix<double>* matrix, Settings* settings);
#ifdef MPI_AVAILABLE
template extern Result gpuSPMultiGPUMPI<double, double>(Matrix<double>* matrix, Settings* settings);
#endif
// FOR COMPILATION -SPARSE-


// FOR COMPILATION -DENSE-
// Calculation: Double | Storage: Float
template extern Result gpuDPSingleGPU<double, float>(Matrix<float>* matrix, Settings* settings);
template extern Result gpuDPMultiGPU<double, float>(Matrix<float>* matrix, Settings* settings);
#ifdef MPI_AVAILABLE
template extern Result gpuDPMultiGPUMPI<double, float>(Matrix<float>* matrix, Settings* settings);
#endif

// Calculation: Double | Storage: Double
template extern Result gpuDPSingleGPU<double, double>(Matrix<double>* matrix, Settings* settings);
template extern Result gpuDPMultiGPU<double, double>(Matrix<double>* matrix, Settings* settings);
#ifdef MPI_AVAILABLE
template extern Result gpuDPMultiGPUMPI<double, double>(Matrix<double>* matrix, Settings* settings);
#endif
// FOR COMPILATION -DENSE-
