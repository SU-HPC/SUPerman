//
// Created by deniz on 5/24/24.
//

#include "GPUKernelWrappers.h"

// COMMON
#include "Permanent.h"
#include "DecomposePerman.h"
#include "Result.h"
#include "Settings.h"
#include "Matrix.h"
#include <stdexcept>
#include "SparseMatrix.h"
#include "hip/hip_runtime.h"

// SP
#include "SparseKernelDefinitions.cuh"
#include "spSingleGPU.cuh"
#include "spMultiGPU.cuh"

// DP
#include "DenseKernelDefinitions.cuh"
#include "dpSingleGPU.cuh"
#include "dpMultiGPU.cuh"

// MPI
#ifdef MPI
#include "spMultiGPUMPI.cuh"
#include "dpMultiGPUMPI.cuh"
#endif


// SPARSE WRAPPERS
template <typename C, typename S>
extern Result gpuSPSingleGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xLocalMShared, spMShared<C, S> > >(Algorithm::XLOCALMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xLocalMGlobal, spNoShared<C, S> > >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> > >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spSingleGPU<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> > >(Algorithm::XSHAREDMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    return result;
}

template <typename C, typename S>
extern Result gpuSPMultiGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xLocalMShared, spMShared<C, S> > >(Algorithm::XLOCALMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xLocalMGlobal, spNoShared<C, S> > >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> > >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, spMultiGPU<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> > >(Algorithm::XSHAREDMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    return result;
}

#ifdef MPI
template <typename C, typename S>
extern Result gpuSPMultiGPUMPI(Matrix<S>* matrix, Settings* settings)
{
    Permanent<C, S>* permanent;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        permanent = new spMultiGPUMPI<C, S, &SparseDefinitions::xLocalMShared, spMShared<C, S> >(Algorithm::XLOCALMSHARED, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        permanent = new spMultiGPUMPI<C, S, &SparseDefinitions::xLocalMGlobal, spNoShared<C, S> >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        permanent = new spMultiGPUMPI<C, S, &SparseDefinitions::xSharedMGlobal, spXShared<C, S> >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        permanent = new spMultiGPUMPI<C, S, &SparseDefinitions::xSharedMShared, spXSharedMShared<C, S> >(Algorithm::XSHAREDMSHARED, matrix, *settings);
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    Result result = permanent->computePermanent();
    delete permanent;
    return result;
}
#endif
// SPARSE WRAPPERS


// DENSE WRAPPERS
template <typename C, typename S>
extern Result gpuDPSingleGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xLocalMShared, dpMShared<C, S> > >(Algorithm::XLOCALMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> > >(Algorithm::XREGISTERMSHARED, matrix, *settings);
            result = permanent->computePermanentRecursively();
            delete permanent;
        #else
            auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> > >(Algorithm::XREGISTERMSHARED, matrix, *settings);
            result = permanent->computePermanentRecursively();
            delete permanent;
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xLocalMGlobal, dpNoShared<C, S> > >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> > >(Algorithm::XREGISTERMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> > >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpSingleGPU<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> > >(Algorithm::XSHAREDMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    return result;
}

template <typename C, typename S>
extern Result gpuDPMultiGPU(Matrix<S>* matrix, Settings* settings)
{
    Result result;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xLocalMShared, dpMShared<C, S> > >(Algorithm::XLOCALMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> > >(Algorithm::XREGISTERMSHARED, matrix, *settings);
            result = permanent->computePermanentRecursively();
            delete permanent;
        #else
            auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> > >(Algorithm::XREGISTERMSHARED, matrix, *settings);
            result = permanent->computePermanentRecursively();
            delete permanent;
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xLocalMGlobal, dpNoShared<C, S> > >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> > >(Algorithm::XREGISTERMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> > >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        auto permanent = new DecomposePerman<C, S, dpMultiGPU<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> > >(Algorithm::XSHAREDMSHARED, matrix, *settings);
        result = permanent->computePermanentRecursively();
        delete permanent;
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    return result;
}

#ifdef MPI
template <typename C, typename S>
extern Result gpuDPMultiGPUMPI(Matrix<S>* matrix, Settings* settings)
{
    Permanent<C, S>* permanent;

    Algorithm selectedAlgorithm = settings->algorithm;
    if (selectedAlgorithm == Algorithm::XLOCALMSHARED)
    {
        permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xLocalMShared, dpMShared<C, S> >(Algorithm::XLOCALMSHARED, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMSHARED)
    {
        #ifdef MAT_SPECIFIC_COMPILATION
            permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMSharedMatSpecificCompilation, dpMShared<C, S> >(Algorithm::XREGISTERMSHARED, matrix, *settings);
        #else
            permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMShared, dpMShared<C, S> >(Algorithm::XREGISTERMSHARED, matrix, *settings);
        #endif
    }
    else if (selectedAlgorithm == Algorithm::XLOCALMGLOBAL)
    {
        permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xLocalMGlobal, dpNoShared<C, S> >(Algorithm::XLOCALMGLOBAL, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XREGISTERMGLOBAL)
    {
        permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xRegisterMGlobal, dpNoShared<C, S> >(Algorithm::XREGISTERMGLOBAL, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMGLOBAL)
    {
        permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xSharedMGlobal, dpXShared<C, S> >(Algorithm::XSHAREDMGLOBAL, matrix, *settings);
    }
    else if (selectedAlgorithm == Algorithm::XSHAREDMSHARED)
    {
        permanent = new dpMultiGPUMPI<C, S, &DenseDefinitions::xSharedMShared, dpXSharedMShared<C, S> >(Algorithm::XSHAREDMSHARED, matrix, *settings);
    }
    else
    {
        throw std::runtime_error("Algorithm you have selected is not included in the available GPU algorithms list.");
    }

    Result result = permanent->computePermanent();
    delete permanent;
    return result;
}
#endif
// DENSE WRAPPERS


// FOR COMPILATION -SPARSE-
// Calculation: Double | Storage: Float
template extern Result gpuSPSingleGPU<double, float>(Matrix<float>* matrix, Settings* settings);
template extern Result gpuSPMultiGPU<double, float>(Matrix<float>* matrix, Settings* settings);
#ifdef MPI
template extern Result gpuSPMultiGPUMPI<double, float>(Matrix<float>* matrix, Settings* settings);
#endif

// Calculation: Double | Storage: Double
template extern Result gpuSPSingleGPU<double, double>(Matrix<double>* matrix, Settings* settings);
template extern Result gpuSPMultiGPU<double, double>(Matrix<double>* matrix, Settings* settings);
#ifdef MPI
template extern Result gpuSPMultiGPUMPI<double, double>(Matrix<double>* matrix, Settings* settings);
#endif
// FOR COMPILATION -SPARSE-


// FOR COMPILATION -DENSE-
// Calculation: Double | Storage: Float
template extern Result gpuDPSingleGPU<double, float>(Matrix<float>* matrix, Settings* settings);
template extern Result gpuDPMultiGPU<double, float>(Matrix<float>* matrix, Settings* settings);
#ifdef MPI
template extern Result gpuDPMultiGPUMPI<double, float>(Matrix<float>* matrix, Settings* settings);
#endif

// Calculation: Double | Storage: Double
template extern Result gpuDPSingleGPU<double, double>(Matrix<double>* matrix, Settings* settings);
template extern Result gpuDPMultiGPU<double, double>(Matrix<double>* matrix, Settings* settings);
#ifdef MPI
template extern Result gpuDPMultiGPUMPI<double, double>(Matrix<double>* matrix, Settings* settings);
#endif
// FOR COMPILATION -DENSE-
